#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <utility>
#include <memory>
#include "utilities.h"

__global__ void kernel(void) {}

int main(int argc, char* argv[]) {
    kernel<<<1, 1>>>();
    std::cout << "Hello world!\n";
    return EXIT_SUCCESS;
}