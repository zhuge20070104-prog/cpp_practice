#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <utility>
#include <memory>
#include "utilities.h"

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(int argc, char* argv[]) {
    int c;
    int *dev_c;
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
    add<<<1, 1>>>(3, 8, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "3 + 8 = " << c << "\n";
    HANDLE_ERROR(hipFree(dev_c));
    return EXIT_SUCCESS;
}