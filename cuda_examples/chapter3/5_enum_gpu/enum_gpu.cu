#include <iostream>
#include <string>
#include <sstream>
#include <utility>
#include <memory>
#include "utilities.h"


int main(int argc, char* argv[]) {
    hipDeviceProp_t prop;
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for(int i=0; i<count; ++i) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        std::cout << "  --- General information of device " << i << "   ---\n";
        std::cout << "Name: " << prop.name << "\n";
        std::cout << "Compute capability: " << prop.major << "." << prop.minor << "\n";
        std::cout << "Clock rate: " << prop.clockRate << "\n";
        std::cout << "Device copy overlap: ";
        if(prop.deviceOverlap) 
            std::cout << "Enabled\n";
        else 
            std::cout << "Disabled\n";
        
        std::cout << "  --- Memory information of device " << i << "   ---\n";
        std::cout << "Total global memory: " << prop.totalGlobalMem << "\n";
        std::cout << "Total const memory: " << prop.totalConstMem << "\n";
        std::cout << "Max mem pitch: " << prop.memPitch << "\n";
        std::cout << "Texture alignment: " << prop.textureAlignment << "\n";

        std::cout << "  --- MP information of device " << i << "   ---\n";
        std::cout << "Multiprocessor count: " << prop.multiProcessorCount << "\n";
        std::cout << "Shared memory per mp: " << prop.sharedMemPerBlock << "\n";
        std::cout << "Register per mp: " << prop.regsPerBlock << "\n";
        std::cout << "Threads in warp: " << prop.warpSize << "\n";
        std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";

        std::cout << "Max thread dimensons: (" << prop.maxThreadsDim[0] << ", " <<
            prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";  
        
        std::cout << "Max grid dimensons: (" << prop.maxGridSize[0] << ", " <<
            prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";
        
        std::cout << "\n";
    }   
    return EXIT_SUCCESS;
}