#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <utility>
#include <memory>
#include "utilities.h"

__device__ int addem(int a, int b) {
    return a+b;
}

__global__ void add(int a, int b, int *c) {
    *c = addem(a, b);
}

int main(int argc, char* argv[]) {
    int c;
    int *dev_c;
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
    add<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "2 + 7 = " << c << "\n";
    HANDLE_ERROR(hipFree(dev_c));
    return EXIT_SUCCESS;
}

